#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <stdio.h>
#include <vector>
#include <math.h>
#include <float.h>
#include "rroi_align_kernel.h"


#define DIVUP(m, n) ((m) / (m) + ((m) % (n) > 0))
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)


// /*
// rroi代码
 template <typename scalar_t>
__global__ void RROIAlignForward(
    const int nthreads,
    const scalar_t* bottom_data,
    const float spatial_scale,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const scalar_t* bottom_rois,
    scalar_t* top_data,
    scalar_t* con_idx_x,
    scalar_t* con_idx_y)
{

    CUDA_KERNEL_LOOP(index, nthreads)
    {
        // +0.5 shift removed
        int imageWidth = width;
        int imageHeight = height;

        // (n, c, ph, pw) is an element in the pooled output
        int n = index;
        int pw = n % pooled_width;
        n /= pooled_width;
        int ph = n % pooled_height;
        n /= pooled_height;
        int c = n % channels;
        n /= channels;

        const scalar_t* offset_bottom_rois = bottom_rois + n * 6; // 标注信息

        int roi_batch_ind = offset_bottom_rois[0];
        scalar_t cx = offset_bottom_rois[1];
        scalar_t cy = offset_bottom_rois[2];
        scalar_t h = offset_bottom_rois[3];
        scalar_t w = offset_bottom_rois[4];
        scalar_t angle = offset_bottom_rois[5]/180.0*3.1415926535;

        //TransformPrepare
        scalar_t roi_pooled_width = pooled_height * w / h;         // 不同的高宽比
        scalar_t dx = -roi_pooled_width/2.0;
        scalar_t dy = -pooled_height/2.0;
        scalar_t Sx = w*spatial_scale/roi_pooled_width;
        scalar_t Sy = h*spatial_scale/pooled_height;
        scalar_t Alpha = cos(angle);
        scalar_t Beta = sin(angle);
        scalar_t Dx = cx*spatial_scale;
        scalar_t Dy = cy*spatial_scale;

        scalar_t M[2][3];                              // 旋转矩阵
        M[0][0] = Alpha*Sx;
        M[0][1] = Beta*Sy;
        M[0][2] = Alpha*Sx*dx+Beta*Sy*dy+Dx;
        M[1][0] = -Beta*Sx;
        M[1][1] = Alpha*Sy;
        M[1][2] = -Beta*Sx*dx+Alpha*Sy*dy+Dy;

        scalar_t P[8];                                 // 求原roi中4个点的坐标8个值
        P[0] = M[0][0]*pw+M[0][1]*ph+M[0][2];
        P[1] = M[1][0]*pw+M[1][1]*ph+M[1][2];
        P[2] = M[0][0]*pw+M[0][1]*(ph+1)+M[0][2];
        P[3] = M[1][0]*pw+M[1][1]*(ph+1)+M[1][2];
        P[4] = M[0][0]*(pw+1)+M[0][1]*ph+M[0][2];
        P[5] = M[1][0]*(pw+1)+M[1][1]*ph+M[1][2];
        P[6] = M[0][0]*(pw+1)+M[0][1]*(ph+1)+M[0][2];
        P[7] = M[1][0]*(pw+1)+M[1][1]*(ph+1)+M[1][2];

        // 求原rroi的中心，并用双线性插值求出f(x,y)
        scalar_t leftMost = (max(round(min(min(P[0],P[2]),min(P[4],P[6]))),0.0));
        scalar_t rightMost= (min(round(max(max(P[0],P[2]),max(P[4],P[6]))),imageWidth-1.0));
        scalar_t topMost= (max(round(min(min(P[1],P[3]),min(P[5],P[7]))),0.0));
        scalar_t bottomMost= (min(round(max(max(P[1],P[3]),max(P[5],P[7]))),imageHeight-1.0));

        const scalar_t* offset_bottom_data = bottom_data + (roi_batch_ind * channels + c) * height * width;

        scalar_t bin_cx = (leftMost + rightMost) / 2.0; // rroi的中心
        scalar_t bin_cy = (topMost + bottomMost) / 2.0;

        const bool in_rroi = pw <= roi_pooled_width;        // 是否在rroi之内
        if (in_rroi){

            int bin_l = (int)floor(bin_cx);
            int bin_r = (int)ceil(bin_cx);
            int bin_t = (int)floor(bin_cy);
            int bin_b = (int)ceil(bin_cy);

            scalar_t lt_value = 0.0;
            if (bin_t > 0 && bin_l > 0 && bin_t < height && bin_l < width)
                lt_value = offset_bottom_data[bin_t * width + bin_l];
            scalar_t rt_value = 0.0;
            if (bin_t > 0 && bin_r > 0 && bin_t < height && bin_r < width)
                rt_value = offset_bottom_data[bin_t * width + bin_r];
            scalar_t lb_value = 0.0;
            if (bin_b > 0 && bin_l > 0 && bin_b < height && bin_l < width)
                lb_value = offset_bottom_data[bin_b * width + bin_l];
            scalar_t rb_value = 0.0;
            if (bin_b > 0 && bin_r > 0 && bin_b < height && bin_r < width)
                rb_value = offset_bottom_data[bin_b * width + bin_r];

            scalar_t rx = bin_cx - floor(bin_cx);
            scalar_t ry = bin_cy - floor(bin_cy);

            scalar_t wlt = (1.0 - rx) * (1.0 - ry);
            scalar_t wrt = rx * (1.0 - ry);
            scalar_t wrb = rx * ry;
            scalar_t wlb = (1.0 - rx) * ry;

            scalar_t inter_val = 0.0;

            inter_val += lt_value * wlt;
            inter_val += rt_value * wrt;
            inter_val += rb_value * wrb;
            inter_val += lb_value * wlb;

            atomicAdd(top_data + index, static_cast<float>(inter_val));
            atomicAdd(con_idx_x + index, static_cast<float>(bin_cx));
            atomicAdd(con_idx_y + index, static_cast<float>(bin_cy));

            //top_data[index] = static_cast<float>(inter_val);
            //con_idx_x[index] = bin_cx;
            //con_idx_y[index] = bin_cy;
        }
        else{
            // float inter_val = 0.0;
            // float bin_cx = 0.0;                        // -2只是为了反向传播时做标记，其他值也是可以的
            // float bin_cy = 0.0;
            // atomicAdd(top_data + index, static_cast<float>(inter_val));     // 可能多个点加了-2
            // atomicAdd(con_idx_x + index, static_cast<float>(bin_cx));
            // atomicAdd(con_idx_y + index, static_cast<float>(bin_cy));
            continue;
        }

    }
}
// 反向传播
template <typename scalar_t>
__global__ void RROIAlignBackward(
    const int nthreads,
    const scalar_t* top_diff,
    const scalar_t* con_idx_x,
    const scalar_t* con_idx_y,
    const int num_rois,
    const float spatial_scale,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    scalar_t* bottom_diff,
    const scalar_t* bottom_rois) {
        CUDA_KERNEL_LOOP(index, nthreads){

        // (n, c, ph, pw) is an element in the pooled output
        // int n = index;
        // //int w = n % width;
        // n /= pooled_width;
        // //int h = n % height;
        // n /= pooled_height;
        // int c = n % channels;
        // n /= channels;

        int n = index;
        int pw = n % pooled_width;
        n /= pooled_width;
        // int ph = n % pooled_height;
        n /= pooled_height;
        int c = n % channels;
        n /= channels;

        const scalar_t* offset_bottom_rois = bottom_rois + n * 6;                    // 第i个rroi
        int roi_batch_ind = offset_bottom_rois[0];
        scalar_t h = offset_bottom_rois[3];
        scalar_t w = offset_bottom_rois[4];
        scalar_t* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;      // 反向梯度的索引

        scalar_t bin_cx = con_idx_x[index];                // 每个rroi中心点的坐标
        scalar_t bin_cy = con_idx_y[index];
        
        // check whether in rroi
        float roi_pooled_width = pooled_height * w / h;         // 不同的高宽比

        const bool not_in_rroi = (pw > roi_pooled_width);    // 可能多个点多次加了-2, 所以不能采用这种方式判断

        if (not_in_rroi){                               // 如果不再rroi内则跳过当前循环，否则就按原来的操作
            continue;
        }
        else{

            scalar_t rx = bin_cx - floor(bin_cx);
            scalar_t ry = bin_cy - floor(bin_cy);

            scalar_t wlt = (1.0 - rx) * (1.0 - ry);
            scalar_t wrt = rx * (1.0 - ry);
            scalar_t wrb = rx * ry;
            scalar_t wlb = (1.0 - rx) * ry;

            int min_x = (int)floor(bin_cx);
            int max_x = (int)ceil(bin_cx);
            int min_y = (int)floor(bin_cy);
            int max_y = (int)ceil(bin_cy);

            scalar_t top_diff_of_bin = top_diff[index];

            scalar_t v1 = wlt * top_diff_of_bin;
            scalar_t v2 = wrt * top_diff_of_bin;
            scalar_t v3 = wrb * top_diff_of_bin;
            scalar_t v4 = wlb * top_diff_of_bin;

            // Atomic add

            if (min_y > 0 && min_x  > 0 && min_y < height - 1 && min_x < width - 1)
                atomicAdd(offset_bottom_diff + min_y * width + min_x, static_cast<float>(v1));          // 多个roi会重复操作
            if (min_y > 0 && max_x < width - 1 && min_y < height - 1 && max_x > 0)
                atomicAdd(offset_bottom_diff + min_y * width + max_x, static_cast<float>(v2));
            if (max_y < height - 1 && max_x < width - 1 && max_y > 0 && max_x > 0)
                atomicAdd(offset_bottom_diff + max_y * width + max_x, static_cast<float>(v3));
            if (max_y < height - 1 && min_x > 0 && max_y > 0 && min_x < width - 1)
                atomicAdd(offset_bottom_diff + max_y * width + min_x, static_cast<float>(v4));

        }
    }
}





int RROIAlignForwardLaucher(
    const at::Tensor& bottom_data, 
    const float spatial_scale, 
    const int num_rois, 
    const int height,
    const int width, 
    const int channels, 
    const int pooled_height,
    const int pooled_width, 
    const at::Tensor& bottom_rois,
    at::Tensor& top_data, 
    at::Tensor& con_idx_x, 
    at::Tensor& con_idx_y, 
    hipStream_t stream)
{
    const int kThreadsPerBlock = 1024;
    const int output_size = num_rois * pooled_height * pooled_width * channels;

    AT_DISPATCH_FLOATING_TYPES(bottom_data.scalar_type(), "RROIAlignForwardLaucher", [&]{
        RROIAlignForward<scalar_t><<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
            output_size, 
            bottom_data.data_ptr<scalar_t>(), 
            spatial_scale, 
            height, 
            width, 
            channels, 
            pooled_height, 
            pooled_width, 
            bottom_rois.data_ptr<scalar_t>(), 
            top_data.data_ptr<scalar_t>(), 
            con_idx_x.data_ptr<scalar_t>(), 
            con_idx_y.data_ptr<scalar_t>());
    });

    THCudaCheck(hipGetLastError());
    return 1;
}

// */





int RROIAlignBackwardLaucher(
    const at::Tensor& top_diff,
    const float spatial_scale,
    const int batch_size,
    const int num_rois,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const at::Tensor& bottom_rois,
    at::Tensor& bottom_diff,
    const at::Tensor& con_idx_x,
    const at::Tensor& con_idx_y,
    hipStream_t stream)
{
    const int kThreadsPerBlock = 1024;
    const int output_size = num_rois * pooled_height * pooled_width * channels;//batch_size * height * width * channels;

    AT_DISPATCH_FLOATING_TYPES(top_diff.scalar_type(), "RROIAlignForward", [&]{
        RROIAlignBackward<scalar_t><<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
      output_size, 
      top_diff.data_ptr<scalar_t>(), 
      con_idx_x.data_ptr<scalar_t>(), 
      con_idx_y.data_ptr<scalar_t>(), 
      num_rois, 
      spatial_scale, 
      height, 
      width, 
      channels, 
      pooled_height,
      pooled_width, 
      bottom_diff.data_ptr<scalar_t>(), 
      bottom_rois.data_ptr<scalar_t>());
    });
    
    THCudaCheck(hipGetLastError());
    return 1;
}
